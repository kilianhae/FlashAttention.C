#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define NEG_INFINITY __int_as_float(0xff800000)

# define d 64
# define B_r 32 // How many rows of Q_i are processed by one threadblock
# define B_c 32 // How many rows of K_i and V_i are processed by one threadblock
# define BK 32 // for now = B_c

// thread - 2nd level tiling
# define TM 4 // How many rows f the attention Matrix S are processed by a single thread
# define TN 4 // How many columns of the attention Matrix S are processed by a single thread

# define CACHE_Q 0

__global__
void flash_tiled(float *out, float* out_l, float *K, float *Q, float* V, float scaling, int batch_stride, int T_r, int T_c)
{
  int tid_x = threadIdx.x;
  int tid_y = threadIdx.y;
  int batch_offset = batch_stride * blockIdx.x;

  /*
  all are fully loaded into shared memory, I think we should adjust this as second step to only loading it in tiles of B_r x 32 
  and iterating the mults over the 32 sized tiles this way we can have a larger d, while keeping occupancy high
  */
  __shared__ float Q_i[B_r][d]; 
  __shared__ float K_j[B_r][B_c];
  __shared__ float V_j[B_r][d];
  
  // attention result
  __shared__ float S_i[B_r][B_c];
  
  // assuming B_c = blockdim.x, within a block, number of tiles a thread has to calculate
  const int num_tiles = d/B_c;
  
  float l_i;
  float m_i;

  assert (B_r == B_c && B_r == blockDim.x && B_r == blockDim.y);
  // assert (num_tiles == 1); // Hack: for now

  // this will be automatucally be put onto registers since very small
  float O_i[num_tiles]; // per register

  // o_per_thread_x, o_per_thread_y is a bit like thread coarsening (each thread takes on multiple elements in loading, and potentially storing)
  for (int t = 0; t < num_tiles; t++) {
    O_i[t] = 0;
  }
  
  // row wise statistics
  for (int t = 0; t < num_tiles; t++) {
    l_i = 0.f;
    m_i = NEG_INFINITY;
  }

  // load Q_i
  for (int t=0; t<num_tiles; t++){
    Q_i[tid_y][t * B_c + tid_x] = Q[batch_offset + (blockIdx.y * B_r + tid_y) * d + t * B_c + tid_x ];
  }
  __syncthreads();

  

  // T_c = seq_len (due to K^T) / B_c, chunk over the d dimension
  // T_c is the number of chunks of K, we iterate over them
  for (int j = 0; j < T_c; j++) {
    S_i[tid_y][tid_x] = 0.f;
    float S_ij = 0.f;
    for (int t=0; t<num_tiles; t++){
      // load K_j and V_j, thread idx, idy loads idy,idx
      // we load a tile
      K_j[tid_y][tid_x] = K[batch_offset + (tid_y + j * B_c) * d  + tid_x + t * B_c]; // not with with r and c

      // TO OPTIMIZE, just loading the V_j for now
      V_j[tid_y][t * B_c + tid_x] = V[batch_offset + (tid_y + j * B_c) * d  + tid_x + t * B_c]; // not with with r and c
      __syncthreads();



      for (int dd=0; dd<B_c; dd++){
        S_ij += Q_i[tid_y][t*B_c+dd] * K_j[tid_x][dd]; // this maybe leads to bank conflicts in the K
      }
      __syncthreads();
    }
    S_i[tid_y][tid_x] += scaling * S_ij;
    __syncthreads();

    float last_m = m_i;
    float m = m_i;
    for (int jj = 0; jj < B_c; jj += 1) {
      if (m < S_i[tid_y][jj]) {
              m = S_i[tid_y][jj];
            }
    }
    __syncthreads();
    m_i = m;
    
    // print all of V
    // 2) renormalize current O
    for (int t = 0; t < num_tiles; t++){
      O_i[t] *= exp(last_m - m);
    }
    // 3) renormalize the sum
    float l = exp(last_m - m) * l_i;

    // 4) compute \exp(Q_iK^T_{j+1} - m^{j+1}) = \exp(S_i-m^{j+1})
    float S_id;
    __syncthreads();
    for (int dd = 0; dd < B_c; dd++) {
      S_id = exp(S_i[tid_y][dd] - m);
      l += S_id;
      for (int t = 0; t < num_tiles; t++){
       // replaced o_y with 1
        O_i[t] += S_id * V_j[dd][t * B_c + tid_x];
      }
    }
    l_i = l;
    __syncthreads();
  }

  // normalize the whole thing by the sum and write to output
  for (int t = 0; t < num_tiles; t++){
    out[batch_offset + (blockIdx.y * B_r + tid_y ) * d + t * B_c + tid_x] = O_i[t] / l_i;
  }
}


__global__
void flash_tiled_coarse(float *out, float* out_l, float *K, float *Q, float* V, float scaling, int batch_stride, int T_r, int T_c, int seq_len)
{
  int tid_x = threadIdx.x;
  int tid_y = threadIdx.y;
  int batch_offset = batch_stride * blockIdx.x;

  /*
  all are fully loaded into shared memory SMEM, I think we should adjust this as second step to only loading it in tiles of B_r x 32 
  and iterating the mults over the 32 sized tiles this way we can have a larger d, while keeping occupancy high
  */
  /*
    NOTE: all are fully loaded into shared memory SMEM, I think we should adjust this as second step to only loading it in tiles of B_r x 32 
    and iterating the mults over the 32 sized tiles this way we can have a larger d, while keeping occupancy high
    */

    // statically define in SMEM and still address it with indices
    //__shared__ float Q_i[B_r][d]; // uncomment only if you want to cache over full d (if CACHE_Q = 1)
    __shared__ float Q_i[B_r][BK]; // if you want to save SMEM loads and keep the full Q loaded then change this to [B_r][d]
    
    __shared__ float K_j[B_c][BK+1]; // reduce SMEM bank conflicts by adding 1 column as K will be loaded transposed!
    __shared__ float V_j[B_c][BK];
    
    // attention result
    __shared__ float S_i[B_r][B_c+1]; // reduce SMEM bank conflicts by adding 1 column (in the naive softmax part)
    const int num_tiles = d/BK; // how many tiles are the computation of the attention is split into

    const uint totalResultsBlocktile = B_r * B_c; // number of results to calculate per block
    const uint numThreadsBlocktile = totalResultsBlocktile / (TM * TN); // number of threads needed
    const int threadId_flat = threadIdx.y * blockDim.x + threadIdx.x; // flattened thread id  (used for coalesced loading of tiles)

    // each thread process one block at position:
    const int threadCol = threadId_flat % (B_c / TN);
    const int threadRow = threadId_flat / (B_c / TN);
        
    float l_i[TM]= {0.0};; // storing the intermediate sum of exponentials per row
    float m_i[TM]; // storing the intermediate max value of the rows
    float last_m[TM]; // storing the last max value of the rows
    float O_i[num_tiles * TN * TM] = {0.0}; // storing the intermediate results of the Outputs (each thread stores a chunk TM x TN per tile)
    
    // reset to min
    for (int ii = 0; ii < TM; ii++) {
        m_i[ii] = -INFINITY;
    }

    //WARNING: due to coalsecing I should probably add a second set of variables for using BK+1
    const uint strideK = numThreadsBlocktile / BK; // 64 / 64 = 1
    const uint innerRowK = threadId_flat / BK; // 0-63 / 64, 0000000000000...0
    const uint innerColK = threadId_flat % BK; // 0-63 % 64, 0123456789101112...63

    int id;
    // load Q_i, UNCOMMENT only if your Q is caching over full d
    const uint innerRowQ = threadId_flat / d; // 0-63 / 64, 0000000000000...0
    const uint innerColQ = threadId_flat % d; // 0-63 % 64, 0123456789012...63
    const uint nr_loads = B_r * d / numThreadsBlocktile;

    for (int t=0; t<nr_loads; t++){
      // need to load block of size B_r x d (64 x 64) with numThreadsBlocktile threads
      // if (blockIdx.y * B_r + innerRowQ) * d + innerColQ + t * numThreadsBlocktile / d
      id = (blockIdx.y * B_r + innerRowQ) * d + innerColQ + t * numThreadsBlocktile;
      // 4 x 4 then this is 5 thus 5/
      if (id < d*seq_len){
        Q_i[innerRowQ][innerColQ + t * numThreadsBlocktile] = Q[batch_offset + id];
      }
      else {
        Q_i[innerRowQ][innerColQ + t * numThreadsBlocktile] = 0.0;
      }
    }

    __syncthreads();

    // scratchpad register for register-tiling (coarsening of the matrix mults)
    float regM[TM] = {0.0};
    float regN[TN] = {0.0};

    for (int j = 0; j < T_c; j++) { // iterate of ver the chunks of K and V
        float threadResults[TM * TN] = {0.0}; // storing the intermediate outputs
        
        for (int t=0; t<num_tiles; t++){
            // load K_j and V_j, thread idx, idy loads idy,idx
            // we load a tile
            for (int i=0; i<B_r; i+=strideK){
                // load Q, K and V in tiles (for now we are loading the full V)
                if (not CACHE_Q){Q_i[innerRowK+i][innerColK] = Q[batch_offset + (innerRowK + blockIdx.y * B_r) * d  + i * d + innerColK + t * B_c];
                } // if you cache Q over whole d then remove this line
                id = (innerRowK + j * B_c) * d + i * d + innerColK + t * B_c;
                if (id < d*seq_len){
                    K_j[innerRowK+i][innerColK] = K[batch_offset + id];
                    //V_j[innerRowK+i][innerColK+t*B_c] = V[batch_offset + id];
                } else {
                    K_j[innerRowK+i][innerColK] = 0.0;
                    //V_j[innerRowK+i][innerColK+t*B_c] = 0.0;
                }
        
            }
            __syncthreads();
        
            for (int dd=0; dd<BK; dd++){ // load elements of Q_i and K_j^T into registers
                for (uint i = 0; i < TM; ++i) {
                    if (CACHE_Q){
                        regM[i] = Q_i[(threadRow * TM + i)][dd+t*BK]; // uncomment if you cache Q over full d
                    } else {
                        regM[i] = Q_i[(threadRow * TM + i)][dd];
                    }
                }
                for (uint i = 0; i < TN; ++i) {
                    regN[i] = K_j[threadCol * TN + i][dd];
                }
                for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
                    for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
                        threadResults[resIdxM * TN + resIdxN] += regM[resIdxM] * regN[resIdxN];
                    }
                }
            }
            __syncthreads();
        }
        

        // store the results in S_i, account for causal masking
        for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
            for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
                    S_i[(threadRow * TM + resIdxM)][threadCol * TN + resIdxN] = threadResults[resIdxM * TN + resIdxN] *scaling;
            }
        }
        __syncthreads();

        for (int i=0;i<TM;++i){
            last_m[i] = m_i[i];
            float m = m_i[i];
            for (int jj = 0; jj < B_c; jj += 1) {
                if (m < S_i[threadRow*TM+i][jj]) {
                    m = S_i[threadRow*TM+i][jj];
                }
            }
            m_i[i] = m;
        }

        // 2) renormalize current O
        if (j > 0) {
            for (int t = 0; t < num_tiles; t++){
                for (int i=0;i<TM;++i){
                    for (int jj=0;jj<TN;++jj){
                        O_i[t*TN*TM + i*TN + jj] *= exp(last_m[i] - m_i[i]);
                    }
                }
            }
        }

        // 3) renormalize the sum l_i
        for (int i=0;i<TM;++i){
            l_i[i] *= exp(last_m[i] - m_i[i]);
        }

        // // 4) compute \exp(Q_iK^T_{j+1} - m^{j+1}) = \exp(S_i-m^{j+1}) // TODO: TO OPTIMIZE
        // for (int dd = 0; dd < B_c; dd++) {
        //     for (int ii = 0; ii < TN; ii++){ 
        //         // calculate new sum and load exp(Attention) weights
        //         //check whether thus is in range or not (if not we set it to 0)
        //         //if (idrow+ii < seq_len && idcol+dd < seq_len){
        //         regM[ii] = exp(S_i[threadRow*TM+ii][dd] - m_i[ii]);
        //         l_i[ii] += regM[ii];
        //     }
        //     for (int t = 0; t < num_tiles; t++){
        //         for (int ii=0;ii<TN;ii++){
        //             for (int jj=0;jj<TM;jj++){ // calculate output elements
        //                 regN[jj] = V_j[dd][t * B_c + threadCol * TN + jj];
        //                 O_i[t*TN*TM + ii*TM + jj] += regM[ii] * regN[jj];
        //             }
        //         }
        //     }
        // __syncthreads();
        // }


        for (int t = 0; t < num_tiles; t++){
            // load V
            __syncthreads();
            for (int i=0; i<B_r; i+=strideK){
                id = (innerRowK + j * B_c) * d + i * d + innerColK + t * B_c;
                if (id < d*seq_len){
                    V_j[innerRowK+i][innerColK] = V[batch_offset + id];
                } else {
                    V_j[innerRowK+i][innerColK] = 0.0;
                }
            }
            __syncthreads();

            for (int dd = 0; dd < B_c; dd++) {
                for (int ii = 0; ii < TN; ii++){
                    regM[ii] = exp(S_i[threadRow*TM+ii][dd] - m_i[ii]);
                    if (t==0){
                        l_i[ii] += regM[ii];
                    }
                    regN[ii] = V_j[dd][threadCol * TN + ii];
                }
                for (int ii=0;ii<TN;ii++){
                    for (int jj=0;jj<TM;jj++){ // calculate output elements
                        regN[jj] = V_j[dd][threadCol * TN + jj];
                        O_i[t*TN*TM + ii*TM + jj] += regM[ii] * regN[jj];
                    }
                }
            }
            __syncthreads();
        }
    }

    // normalize by the output sum and write to out matrix
    for (int t = 0; t < num_tiles; t++){
        for (int ii=0;ii<TM;ii++){
            for (int jj=0;jj<TN;jj++){
                if(blockIdx.y*B_r+threadRow*TM+ii < seq_len){
                    out[batch_offset + (blockIdx.y * B_r + threadRow*TM + ii) * d + t * B_c + threadCol*TN + jj] = O_i[t*TN*TM+ii*TM+jj] / l_i[ii];
                }
            }
        } 
    }
}



__global__
void flash_tiled_coarse_causal(float *out, float* out_l, float *K, float *Q, float* V, float scaling, int batch_stride, int T_r, int T_c, int seq_len)
{
  int tid_x = threadIdx.x;
  int tid_y = threadIdx.y;
  int batch_offset = batch_stride * blockIdx.x;

  /*
  all are fully loaded into shared memory SMEM, I think we should adjust this as second step to only loading it in tiles of B_r x 32 
  and iterating the mults over the 32 sized tiles this way we can have a larger d, while keeping occupancy high
  */
  /*
    NOTE: all are fully loaded into shared memory SMEM, I think we should adjust this as second step to only loading it in tiles of B_r x 32 
    and iterating the mults over the 32 sized tiles this way we can have a larger d, while keeping occupancy high
    */

    // statically define in SMEM and still address it with indices
    //__shared__ float Q_i[B_r][d]; // uncomment only if you want to cache over full d (if CACHE_Q = 1)
    __shared__ float Q_i[B_r][BK]; // if you want to save SMEM loads and keep the full Q loaded then change this to [B_r][d]
    
    __shared__ float K_j[B_c][BK+1]; // reduce SMEM bank conflicts by adding 1 column as K will be loaded transposed!
    __shared__ float V_j[B_c][BK];
    
    // attention result
    __shared__ float S_i[B_r][B_c+1]; // reduce SMEM bank conflicts by adding 1 column (in the naive softmax part)
    const int num_tiles = d/BK; // how many tiles are the computation of the attention is split into

    const uint totalResultsBlocktile = B_r * B_c; // number of results to calculate per block
    const uint numThreadsBlocktile = totalResultsBlocktile / (TM * TN); // number of threads needed
    const int threadId_flat = threadIdx.y * blockDim.x + threadIdx.x; // flattened thread id  (used for coalesced loading of tiles)

    // each thread process one block at position:
    const int threadCol = threadId_flat % (B_c / TN);
    const int threadRow = threadId_flat / (B_c / TN);
        
    float l_i[TM]= {0.0};; // storing the intermediate sum of exponentials per row
    float m_i[TM]; // storing the intermediate max value of the rows
    float last_m[TM]; // storing the last max value of the rows
    float O_i[num_tiles * TN * TM] = {0.0}; // storing the intermediate results of the Outputs (each thread stores a chunk TM x TN per tile)
    
    // reset to min
    for (int ii = 0; ii < TM; ii++) {
        m_i[ii] = -INFINITY;
    }

    //WARNING: due to coalsecing I should probably add a second set of variables for using BK+1
    const uint strideK = numThreadsBlocktile / BK; // 64 / 64 = 1
    const uint innerRowK = threadId_flat / BK; // 0-63 / 64, 0000000000000...0
    const uint innerColK = threadId_flat % BK; // 0-63 % 64, 0123456789101112...63

    int id;
    // load Q_i, UNCOMMENT only if your Q is caching over full d
    const uint innerRowQ = threadId_flat / d; // 0-63 / 64, 0000000000000...0
    const uint innerColQ = threadId_flat % d; // 0-63 % 64, 0123456789012...63
    const uint nr_loads = B_r * d / numThreadsBlocktile;

    for (int t=0; t<nr_loads; t++){
      // need to load block of size B_r x d (64 x 64) with numThreadsBlocktile threads
      // if (blockIdx.y * B_r + innerRowQ) * d + innerColQ + t * numThreadsBlocktile / d
      id = (blockIdx.y * B_r + innerRowQ) * d + innerColQ + t * numThreadsBlocktile;
      // 4 x 4 then this is 5 thus 5/
      if (id < d*seq_len){
        Q_i[innerRowQ][innerColQ + t * numThreadsBlocktile] = Q[batch_offset + id];
      }
      else {
        Q_i[innerRowQ][innerColQ + t * numThreadsBlocktile] = 0.0;
      }
    }

    __syncthreads();

    // scratchpad register for register-tiling (coarsening of the matrix mults)
    float regM[TM] = {0.0};
    float regN[TN] = {0.0};

    for (int j = 0; j < T_c && j <= blockIdx.y ; j++) { // iterate of ver the chunks of K and V
        float threadResults[TM * TN] = {0.0}; // storing the intermediate outputs
        
        for (int t=0; t<num_tiles; t++){
            // load K_j and V_j, thread idx, idy loads idy,idx
            // we load a tile
            for (int i=0; i<B_r; i+=strideK){
                // load Q, K and V in tiles (for now we are loading the full V)
                if (not CACHE_Q){Q_i[innerRowK+i][innerColK] = Q[batch_offset + (innerRowK + blockIdx.y * B_r) * d  + i * d + innerColK + t * B_c];
                } // if you cache Q over whole d then remove this line
                id = (innerRowK + j * B_c) * d + i * d + innerColK + t * B_c;
                if (id < d*seq_len){
                    K_j[innerRowK+i][innerColK] = K[batch_offset + id];
                    //V_j[innerRowK+i][innerColK+t*B_c] = V[batch_offset + id];
                } else {
                    K_j[innerRowK+i][innerColK] = 0.0;
                    //V_j[innerRowK+i][innerColK+t*B_c] = 0.0;
                }
        
            }
            __syncthreads();
        
            for (int dd=0; dd<BK; dd++){ // load elements of Q_i and K_j^T into registers
                for (uint i = 0; i < TM; ++i) {
                    if (CACHE_Q){
                        regM[i] = Q_i[(threadRow * TM + i)][dd+t*BK]; // uncomment if you cache Q over full d
                    } else {
                        regM[i] = Q_i[(threadRow * TM + i)][dd];
                    }
                }
                for (uint i = 0; i < TN; ++i) {
                    regN[i] = K_j[threadCol * TN + i][dd];
                }
                for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
                    for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
                        threadResults[resIdxM * TN + resIdxN] += regM[resIdxM] * regN[resIdxN];
                    }
                }
            }
            __syncthreads();
        }
        

        // store the results in S_i, account for causal masking
        for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
            for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
                if (j*B_c + threadCol * TN + resIdxN <= blockIdx.y * B_r + threadRow * TM + resIdxM){
                    S_i[(threadRow * TM + resIdxM)][threadCol * TN + resIdxN] = threadResults[resIdxM * TN + resIdxN] *scaling;
                } else {
                    S_i[(threadRow * TM + resIdxM)][threadCol * TN + resIdxN] = -INFINITY;
                }      
            }
        }
        __syncthreads();

        for (int i=0;i<TM;++i){
            last_m[i] = m_i[i];
            float m = m_i[i];
            for (int jj = 0; jj < B_c; jj += 1) {
                if (m < S_i[threadRow*TM+i][jj]) {
                    m = S_i[threadRow*TM+i][jj];
                }
            }
            m_i[i] = m;
        }

        // 2) renormalize current O
        if (j > 0) {
            for (int t = 0; t < num_tiles; t++){
                for (int i=0;i<TM;++i){
                    for (int jj=0;jj<TN;++jj){
                        O_i[t*TN*TM + i*TN + jj] *= exp(last_m[i] - m_i[i]);
                    }
                }
            }
        }

        // 3) renormalize the sum l_i
        for (int i=0;i<TM;++i){
            l_i[i] *= exp(last_m[i] - m_i[i]);
        }

        // // 4) compute \exp(Q_iK^T_{j+1} - m^{j+1}) = \exp(S_i-m^{j+1}) // TODO: TO OPTIMIZE
        // for (int dd = 0; dd < B_c; dd++) {
        //     for (int ii = 0; ii < TN; ii++){ 
        //         // calculate new sum and load exp(Attention) weights
        //         //check whether thus is in range or not (if not we set it to 0)
        //         //if (idrow+ii < seq_len && idcol+dd < seq_len){
        //         regM[ii] = exp(S_i[threadRow*TM+ii][dd] - m_i[ii]);
        //         l_i[ii] += regM[ii];
        //     }
        //     for (int t = 0; t < num_tiles; t++){
        //         for (int ii=0;ii<TN;ii++){
        //             for (int jj=0;jj<TM;jj++){ // calculate output elements
        //                 regN[jj] = V_j[dd][t * B_c + threadCol * TN + jj];
        //                 O_i[t*TN*TM + ii*TM + jj] += regM[ii] * regN[jj];
        //             }
        //         }
        //     }
        // __syncthreads();
        // }


        for (int t = 0; t < num_tiles; t++){
            // load V
            __syncthreads();
            for (int i=0; i<B_r; i+=strideK){
                id = (innerRowK + j * B_c) * d + i * d + innerColK + t * B_c;
                if (id < d*seq_len){
                    V_j[innerRowK+i][innerColK] = V[batch_offset + id];
                } else {
                    V_j[innerRowK+i][innerColK] = 0.0;
                }
            }
            __syncthreads();

            for (int dd = 0; dd < B_c; dd++) {
                for (int ii = 0; ii < TN; ii++){
                    regM[ii] = exp(S_i[threadRow*TM+ii][dd] - m_i[ii]);
                    if (t==0){
                        l_i[ii] += regM[ii];
                    }
                    regN[ii] = V_j[dd][threadCol * TN + ii];
                }
                for (int ii=0;ii<TN;ii++){
                    for (int jj=0;jj<TM;jj++){ // calculate output elements
                        regN[jj] = V_j[dd][threadCol * TN + jj];
                        O_i[t*TN*TM + ii*TM + jj] += regM[ii] * regN[jj];
                    }
                }
            }
            __syncthreads();
        }
    }

    // normalize by the output sum and write to out matrix
    for (int t = 0; t < num_tiles; t++){
        for (int ii=0;ii<TM;ii++){
            for (int jj=0;jj<TN;jj++){
                if(blockIdx.y*B_r+threadRow*TM+ii < seq_len){
                    out[batch_offset + (blockIdx.y * B_r + threadRow*TM + ii) * d + t * B_c + threadCol*TN + jj] = O_i[t*TN*TM+ii*TM+jj] / l_i[ii];
                }
            }
        } 
    }
}



void run_flash_tiled(torch::Tensor O, torch::Tensor O_l, torch::Tensor K_d, torch::Tensor Q_d, torch::Tensor V_d, int batch_size, int seq_len) {
  dim3 blockDim(B_r, B_c);
  dim3 gridDim(batch_size,  (seq_len+B_r-1)/B_r);
  flash_tiled<<<gridDim, blockDim>>>(O.data_ptr<float>(), O_l.data_ptr<float>(), K_d.data_ptr<float>(), Q_d.data_ptr<float>(), V_d.data_ptr<float>(), (float) 1.0, (int) seq_len * d, (int) seq_len/B_r, (int) seq_len/B_c);
  hipDeviceSynchronize();
}

void run_flash_tiled_coarse(torch::Tensor O, torch::Tensor O_l, torch::Tensor K_d, torch::Tensor Q_d, torch::Tensor V_d, int batch_size, int seq_len) {
  dim3 blockDim(B_r/TN, B_c/TM);
  dim3 gridDim(batch_size, (seq_len+B_r-1)/B_r);
  flash_tiled_coarse<<<gridDim, blockDim>>>(O.data_ptr<float>(), O_l.data_ptr<float>(), K_d.data_ptr<float>(), Q_d.data_ptr<float>(), V_d.data_ptr<float>(), (float) 1.0, (int) seq_len * d, (int) (seq_len+B_r-1)/B_r, (int) (seq_len+B_c-1)/B_c, seq_len);
  hipDeviceSynchronize();
}

void run_flash_tiled_coarse_causal(torch::Tensor O, torch::Tensor O_l, torch::Tensor K_d, torch::Tensor Q_d, torch::Tensor V_d, int batch_size, int seq_len) {
  dim3 blockDim(B_r/TN, B_c/TM);
  dim3 gridDim(batch_size, (seq_len+B_r-1)/B_r);
  flash_tiled_coarse_causal<<<gridDim, blockDim>>>(O.data_ptr<float>(), O_l.data_ptr<float>(), K_d.data_ptr<float>(), Q_d.data_ptr<float>(), V_d.data_ptr<float>(), (float) 1.0, (int) seq_len * d, (int) (seq_len+B_r-1)/B_r, (int) (seq_len+B_c-1)/B_c, seq_len);
  hipDeviceSynchronize();
}
torch::Tensor forward(torch::Tensor Q_d, torch::Tensor K_d, torch::Tensor V_d, bool causal) {
  int batch_size = Q_d.size(0);
  int seq_len = Q_d.size(1);
  assert (Q_d.size(2) == d);

  torch::Tensor O = torch::zeros({batch_size, seq_len, d}, torch::kCUDA);
  torch::Tensor O_l = torch::zeros({batch_size, seq_len}, torch::kCUDA);
  if (causal){
    run_flash_tiled_coarse_causal(O, O_l, K_d, Q_d, V_d, batch_size, seq_len);
  }
  else{
    run_flash_tiled_coarse(O, O_l, K_d, Q_d, V_d, batch_size, seq_len);
  }
  return O;
}