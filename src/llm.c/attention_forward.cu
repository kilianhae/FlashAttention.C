#include "hip/hip_runtime.h"
/*
Kernels for attention forward pass.

Compile example:
nvcc -O3 --use_fast_math attention_forward.cu -o attention_forward -lcublas

version 1 is naive port from CPU code to kernel, parallelize over batch, time, heads only
./attention_forward 1

version 2 is a naive implementation of flash attention, taken, adapted from
https://github.com/tspeterkim/flash-attention-minimal
and with help from
https://github.com/leloykun/flash-hyperbolic-attention-minimal
sadly, this flash attention version seems about 3X slower than the naive version
./attention_forward 2

version 3 is a cuBLAS + softmax version, similar to the PyTorch implementation
cuBLAS is used both to calculate the QK^T and the final weighted sum
the softmax is calculated using a custom, efficient kernel as well
this turns out to be ~20X faster than (1) nice
./attention_forward 3

version 4 is a further optimized kernel that fuses the scale operation,
uses a directly autoregressive softmax, and uses the online softmax algorithm.
./attention_forward 4
*/

#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>
#include <assert.h>
#include <float.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "common.h"

// ----------------------------------------------------------------------------
// CUDA setup

static hipblasHandle_t cublas_handle;

// ----------------------------------------------------------------------------
// CPU code reference
double getTimeStamp() {
    struct timeval tv;
    gettimeofday( &tv, NULL );
    return (double) tv.tv_usec/1000000 + tv.tv_sec;
}


void attention_forward_cpu(float* out, float* preatt, float* att,
                       const float* inp,
                       int B, int T, int C, int NH) {
    // input is (B, T, 3C) Q,K,V
    // preatt, att are (B, NH, T, T)
    // output is (B, T, C)
    int C3 = C*3;
    int hs = C / NH; // head size
    float scale = 1.0 / sqrtf(hs);

    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            for (int h = 0; h < NH; h++) {
                const float* query_t = inp + b * T * C3 + t * C3 + h * hs;
                float* preatt_bth = preatt + b*NH*T*T + h*T*T + t*T;
                float* att_bth = att + b*NH*T*T + h*T*T + t*T;

                // pass 1: calculate query dot key and maxval
                float maxval = -10000.0f; // TODO something better
                for (int t2 = 0; t2 <= t; t2++) {
                    const float* key_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C; // +C because it's key

                    // (query_t) dot (key_t2)
                    float val = 0.0f;
                    for (int i = 0; i < hs; i++) {
                        val += query_t[i] * key_t2[i];
                    }
                    val *= scale;
                    if (val > maxval) {
                        maxval = val;
                    }

                    preatt_bth[t2] = val;
                }
                // pad with -INFINITY outside of autoregressive region for debugging comparisons
                for (int t2 = t+1; t2 < T; t2++) {
                    preatt_bth[t2] = -INFINITY;
                }

                // pass 2: calculate the exp and keep track of sum
                float expsum = 0.0f;
                for (int t2 = 0; t2 <= t; t2++) {
                    float expv = expf(preatt_bth[t2] - maxval);
                    expsum += expv;
                    att_bth[t2] = expv;
                }
                float expsum_inv = expsum == 0.0f ? 0.0f : 1.0f / expsum;

                // pass 3: normalize to get the softmax
                for (int t2 = 0; t2 < T; t2++) {
                    if (t2 <= t) {
                        att_bth[t2] *= expsum_inv;
                    } else {
                        // causal attention mask. not strictly necessary to set to zero here
                        // only doing this explicitly for debugging and checking to PyTorch
                        att_bth[t2] = 0.0f;
                    }
                }

                // pass 4: accumulate weighted values into the output of attention
                float* out_bth = out + b * T * C + t * C + h * hs;
                for (int i = 0; i < hs; i++) { out_bth[i] = 0.0f; }
                for (int t2 = 0; t2 <= t; t2++) {
                    const float* value_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C*2; // +C*2 because it's value
                    float att_btht2 = att_bth[t2];
                    for (int i = 0; i < hs; i++) {
                        out_bth[i] += att_btht2 * value_t2[i];
                    }
                }
            }
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

__global__ void attention_query_key_kernel1(float* preatt, const float* inp,
                                           int B, int T, int C, int NH) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = B * NH * T * T;

    if (idx < total_threads) {
        int t2 = idx % T;
        int t = (idx / T) % T;
        if (t2 > t) {
            // autoregressive mask
            preatt[idx] = -INFINITY;
            return;
        }
        int h = (idx / (T * T)) % NH;
        int b = idx / (NH * T * T);

        int C3 = C*3;
        int hs = C / NH; // head size
        const float* query_t = inp + b * T * C3 + t * C3 + h * hs;
        const float* key_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C; // +C because it's key

        // (query_t) dot (key_t2)
        float val = 0.0f;
        for (int i = 0; i < hs; i++) {
            val += query_t[i] * key_t2[i];
        }
        val *= 1.0 / sqrtf(hs);

        preatt[idx] = val;
    }
}

__global__ void attention_softmax_kernel1(float* att, const float* preatt,
                                         int B, int T, int NH) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = B * T * NH;

    if (idx < total_threads) {
        int h = idx % NH;
        int t = (idx / NH) % T;
        int b = idx / (NH * T);

        const float* preatt_bth = preatt + b*NH*T*T + h*T*T + t*T;
        float* att_bth = att + b*NH*T*T + h*T*T + t*T;

        // find maxval
        float maxval = -10000.0f; // TODO something better
        for (int t2 = 0; t2 <= t; t2++) {
            if (preatt_bth[t2] > maxval) {
                maxval = preatt_bth[t2];
            }
        }

        // calculate the exp and keep track of sum
        float expsum = 0.0f;
        for (int t2 = 0; t2 <= t; t2++) {
            float expv = expf(preatt_bth[t2] - maxval);
            expsum += expv;
            att_bth[t2] = expv;
        }
        float expsum_inv = expsum == 0.0f ? 0.0f : 1.0f / expsum;

        // normalize to get the softmax
        for (int t2 = 0; t2 < T; t2++) {
            if (t2 <= t) {
                att_bth[t2] *= expsum_inv;
            } else {
                // causal attention mask. not strictly necessary to set to zero here
                // only doing this explicitly for debugging and checking to PyTorch
                att_bth[t2] = 0.0f;
            }
        }
    }
}

// warp-level reduction for finding the maximum value
__device__ float warpReduceMax(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val = fmaxf(val, __shfl_down_sync(0xFFFFFFFF, val, offset));
    }
    return val;
}

// warp-level reduction for summing values
__device__ float warpReduceSum(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    }
    return val;
}

__global__ void softmax_forward_kernel4(float* out, const float* inp, int N, int C) {
    // out is (N, C) just like inp. Each row of inp will get softmaxed.
    // same as kernel3, but can handle any block size (multiple of 32)
    // each row of C elements is handled by block_size threads
    // furthermore, each block_size threads get executed in warps of 32 threads

    // special reduction operations warpReduceMax/warpReduceSum are used for intra-warp reductions
    // shared memory is used for inter-warp reduction
    extern __shared__ float shared[];
    int idx = blockIdx.x;
    int tid = threadIdx.x;
    int warpId = threadIdx.x / 32; // warp index within a block
    int laneId = threadIdx.x % 32; // thread index within a warp

    // the number of warps per block. recall that blockDim.x is block_size
    int warpsPerBlock = blockDim.x / 32;

    // shared[] must be allocated to have 2 * warpsPerBlock elements
    // first half for max values, the second half for sum values
    float* maxvals = shared;
    float* sumvals = &shared[warpsPerBlock];

    // one row of inp, i.e. inp[idx, :] of shape (C,)
    const float* x = inp + idx * C;

    // first, thread coarsening by directly accessing global memory in series
    float maxval = -INFINITY;
    for (int i = tid; i < C; i += blockDim.x) {
        maxval = fmaxf(maxval, x[i]);
    }
    // now within-warp reductions for maxval
    maxval = warpReduceMax(maxval);

    // the 0th thread of each warp writes the maxval of that warp to shared memory
    if (laneId == 0) maxvals[warpId] = maxval;
    __syncthreads();

    // now the 0th thread reduces the maxvals in shared memory, i.e. across warps
    if (tid == 0) {
        float val = maxvals[tid];
        for (int i = 1; i < warpsPerBlock; i++) {
            val = fmaxf(val, maxvals[i]);
        }
        // store the final max in the first position
        maxvals[0] = val;
    }
    __syncthreads();
    // broadcast the max to all threads
    float offset = maxvals[0];

    // compute expf and write the result to global memory
    for (int i = tid; i < C; i += blockDim.x) {
        // subtract max for numerical stability
        out[idx * C + i] = expf(x[i] - offset);
    }

    // okay now we calculated exp(x - max(x))
    // step 2: sum all the values and divide by the sum

    // thread coarsening for sum
    x = out + idx * C;
    float sumval = 0.0f;
    for (int i = tid; i < C; i += blockDim.x) {
        sumval += x[i];
    }
    // within-warp reduction for sumval
    sumval = warpReduceSum(sumval);

    // write sumval to shared memory
    if (laneId == 0) sumvals[warpId] = sumval;
    __syncthreads();

    // inter-thread reduction of sum
    if (tid == 0) {
        float val = sumvals[tid];
        for (int i = 1; i < warpsPerBlock; ++i) {
            val += sumvals[i];
        }
        sumvals[0] = val;
    }
    __syncthreads();
    // broadcast the sum to all threads
    float sum = sumvals[0];

    // divide the whole row by the sum
    for (int i = tid; i < C; i += blockDim.x) {
        out[idx * C + i] = x[i] / sum;
    }
}


__device__ float& vec_at(float4& vec, int index) {
    return reinterpret_cast<float*>(&vec)[index];
}

__device__ float vec_at(const float4& vec, int index) {
    return reinterpret_cast<const float*>(&vec)[index];
}

__global__ void softmax_forward_kernel5(float* out, float inv_temperature, const float* inp, int N, int T) {
    // inp, out shape: (N, T, T), where N = B * NH
    // fuses the multiplication by scale inside attention
    // directly autoregressive, so we only compute the lower triangular part
    // uses the online softmax algorithm
    assert(T % 4  == 0);
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
    if(idx >= N * T) {
        return;
    }
    int own_pos = idx % T;
    int pos_by_4 = own_pos / 4;

    // one row of inp, i.e. inp[idx, :] of shape (T,)
    const float* x = inp + idx * T;

    // not INF, so we don't get NaNs accidentally when subtracting two values.
    float maxval = -FLT_MAX;
    float sumval = 0.0f;

    const float4* x_vec = reinterpret_cast<const float4*>(x);
    for (int i = warp.thread_rank(); i < pos_by_4; i += warp.size()) {
        float4 v = x_vec[i];
        float old_maxval = maxval;
        for(int k = 0; k < 4; ++k) {
            maxval = fmaxf(maxval, vec_at(v, k));
        }
        sumval *= expf(inv_temperature * (old_maxval - maxval));
        for(int k = 0; k < 4; ++k) {
            sumval += expf(inv_temperature * (vec_at(v, k) - maxval));
        }
    }

    if(4*pos_by_4 + warp.thread_rank() <= own_pos) {
        float old_maxval = maxval;
        maxval = fmaxf(maxval, x[4*pos_by_4 + warp.thread_rank()]);
        sumval *= expf(inv_temperature * (old_maxval - maxval));
        sumval += expf(inv_temperature * (x[4*pos_by_4 + warp.thread_rank()] - maxval));
    }

    float global_maxval = cg::reduce(warp, maxval, cg::greater<float>{});
    sumval *= expf(inv_temperature * (maxval - global_maxval));

    float sum = cg::reduce(warp, sumval, cg::plus<float>{});
    float norm = 1.f / sum;

    // divide the whole row by the sum
    for (int i = warp.thread_rank(); i <= own_pos; i += warp.size()) {
        // recalculation is faster than doing the round-trip through memory.
        float ev = expf(inv_temperature * (__ldcs(x + i) - global_maxval));
        __stcs(out + idx * T + i, ev * norm);
    }
}


__global__ void attention_value_kernel1(float* out, const float* att, const float* inp,
                                       int B, int T, int C, int NH) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = B * T * NH;

    if (idx < total_threads) {
        int h = idx % NH;
        int t = (idx / NH) % T;
        int b = idx / (NH * T);

        int C3 = C*3;
        int hs = C / NH; // head size

        float* out_bth = out + b * T * C + t * C + h * hs;
        const float* att_bth = att + b*NH*T*T + h*T*T + t*T;

        for (int i = 0; i < hs; i++) { out_bth[i] = 0.0f; }
        for (int t2 = 0; t2 <= t; t2++) {
           const  float* value_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C*2; // +C*2 because it's value
            float att_btht2 = att_bth[t2];
            for (int i = 0; i < hs; i++) {
                out_bth[i] += att_btht2 * value_t2[i];
            }
        }
    }
}

__global__
void attention_forward_kernel2(
    const float* Q,
    const float* K,
    const float* V,
    const int N,
    const int d,
    const int Tc,
    const int Tr,
    const int Bc,
    const int Br,
    const float softmax_scale,
    float* l,
    float* m,
    float* O
) {
    int tx = threadIdx.x;
    int bx = blockIdx.x; int by = blockIdx.y;  // batch and head index

    // Offset into Q,K,V,O,l,m - different for each batch and head
    int qkv_offset = (bx * gridDim.y * N * d) + (by * N * d);  // gridDim.y = nh
    int lm_offset = (bx * gridDim.y * N) + (by * N);  // offset for l and m

    // Define SRAM for Q,K,V,S
    extern __shared__ float sram[];
    int tile_size = Bc * d;  // size of Qi, Kj, Vj
    float* Qi = sram;
    float* Kj = &sram[tile_size];
    float* Vj = &sram[tile_size * 2];
    float* S = &sram[tile_size * 3];

    for (int j = 0; j < Tc; j++) {

        // Load Kj, Vj to SRAM
        for (int x = 0; x < d; x++) {
            Kj[(tx * d) + x] = K[qkv_offset + (tile_size * j) + (tx * d) + x];
            Vj[(tx * d) + x] = V[qkv_offset + (tile_size * j) + (tx * d) + x];
        }
        __syncthreads();  // such that the inner loop can use the correct Kj, Vj

        for (int i = 0; i < Tr; i++)  {
            // if past the end of the sequence, break
            if (i * Br + tx >= N) {
                break;
            }

            // Load Qi to SRAM, l and m to registers
            for (int x = 0; x < d; x++) {
                Qi[(tx * d) + x] = Q[qkv_offset + (tile_size * i) + (tx * d) + x];
            }
            float row_m_prev = m[lm_offset + (Br * i) + tx];
            float row_l_prev = l[lm_offset + (Br * i) + tx];

            // S = QK^T, row_m = rowmax(S)
            // S[tx][y] = Sum_{x = 0}^{d-1} {Qi[tx][x] * Kj[y][x]}
            // row_m = Max_{y = 0}^{Bc-1} S[tx][y]
            // with causal masking
            float row_m = -INFINITY;
            for (int y = 0; y < Bc; y++) {
                if (j * Bc + y >= N) {
                    break;
                }
                float sum = 0;
                for (int x = 0; x < d; x++) {
                    sum += Qi[(tx * d) + x] * Kj[(y * d) + x];
                }
                sum *= softmax_scale;
                if (i * Br + tx < j * Bc + y)
                    sum = -INFINITY;
                S[(Bc * tx) + y] = sum;

                if (sum > row_m)
                    row_m = sum;
            }

            // implement softmax with causal masking
            // P = exp(S - row_m), row_l = rowsum(P)
            // P[tx][y] = exp(S[tx][y] - row_m)
            float row_l = 0;
            for (int y = 0; y < Bc; y++) {
                if (j * Bc + y >= N) {
                    break;
                }
                if (i * Br + tx < j * Bc + y)
                    S[(Bc * tx) + y] = 0;
                else
                    S[(Bc * tx) + y] = __expf(S[(Bc * tx) + y] - row_m);
                row_l += S[(Bc * tx) + y];
            }

            // Compute new m and l
            float row_m_new = max(row_m_prev, row_m);
            float row_l_new = (__expf(row_m_prev - row_m_new) * row_l_prev) + (__expf(row_m - row_m_new) * row_l);

            // Write O, l, m to HBM
            for (int x = 0; x < d; x++) {
                float pv = 0;  // Pij * Vj
                for (int y = 0; y < Bc; y++) {
                    if (j * Bc + y >= N) {
                        break;
                    }
                    pv += S[(Bc * tx) + y] * Vj[(y * d) + x];
                }
                O[qkv_offset + (tile_size * i) + (tx * d) + x] = (1 / row_l_new) \
                    * ((row_l_prev * __expf(row_m_prev - row_m_new) * O[qkv_offset + (tile_size * i) + (tx * d) + x]) \
                    + (__expf(row_m - row_m_new) * pv));
            }
            m[lm_offset + (Br * i) + tx] = row_m_new;
            l[lm_offset + (Br * i) + tx] = row_l_new;
        }
        __syncthreads();  // otherwise, thread can use the wrong Kj, Vj in inner loop
    }
}

__global__ void permute_kernel(float* q, float* k, float* v,
                               const float* inp,
                               int B, int N, int NH, int d) {
    // okay so now, this kernel wants Q,K,V to all be of shape (B, NH, N, d)
    // but instead, we have a single tensor QKV (inp) of shape (B, N, 3, NH, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Q[b][nh_][n][d_] = inp[b][n][0][nh_][d_]

    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int inp_idx = \
            (b * N * 3 * NH * d)
            +   (n * 3 * NH * d)
            +       (0 * NH * d)
            +          (nh_ * d)
            +                d_;

        q[idx] = inp[inp_idx];
        k[idx] = inp[inp_idx + NH * d];
        v[idx] = inp[inp_idx + 2 * (NH * d)];
    }
}

__global__ void unpermute_kernel(const float* inp, float *out, int B, int N, int NH, int d) {
   // out has shape (B, nh, N, d) but we need to unpermute it to (B, N, nh, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // out[b][n][nh_][d_] <- inp[b][nh_][n][d_]
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int other_idx = (b * NH * N * d) + (n * NH * d) + (nh_ * d) + d_;
        out[other_idx] = inp[idx];
    }
}

__global__ void scale_kernel(float* inp, float scale, int B, int NH, int T) {
    // scales the pre-softmax attention scores by scale
    // and sets the autoregressive locations to -INFINITY
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < B * NH * T * T) {
        int rest = idx % (NH * T * T);
        rest = rest % (T * T);
        int t2 = rest / T;
        int t = rest % T;
        if (t > t2) {
            inp[idx] = -INFINITY;
        } else {
            inp[idx] *= scale;
        }
    }
}

// direct translation of the CPU kernel. Each warp handles ont (b, h, t) combination.
// The important changes compared to the CPU version:
//  - each inner loop is handled by a warp
//  - don't write non-autoregressive parts
//  - reordered the last loops so that we can do all writing in the outer loop.
__global__ void attention_forward_fused1(float* out, float* preatt, float* att,
                                         const float* inp,
                                         int B, int T, int C, int NH) {
    // input is (B, T, 3C) Q,K,V
    // preatt, att are (B, NH, T, T)
    // output is (B, T, C)
    int C3 = C*3;
    int hs = C / NH; // head size
    float scale = 1.0 / sqrtf(hs);

    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int t = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
    int h = blockIdx.y;
    int b = blockIdx.z;

    if(t >= T) return;

    const float* query_t = inp + b * T * C3 + t * C3 + h * hs;
    float* preatt_bth = preatt + b*NH*T*T + h*T*T + t*T;
    float* att_bth = att + b*NH*T*T + h*T*T + t*T;

    // pass 1: calculate query dot key and maxval
    float maxval = -INFINITY;
    for (int t2 = 0; t2 <= t; t2++) {
        const float* key_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C; // +C because it's key

        // (query_t) dot (key_t2)
        float val = 0.0f;
        for (int i = warp.thread_rank(); i < hs; i += warp.size()) {
            val += query_t[i] * key_t2[i];
        }
        val = cg::reduce(warp, val, cg::plus<float>{});
        val *= scale;
        maxval = max(maxval, val);
        if(warp.thread_rank() == 0) {
            preatt_bth[t2] = val;
        }
    }

    // pass 2: calculate the exp and keep track of sum
    float expsum = 0.0f;
    for (int t2 = warp.thread_rank(); t2 <= t; t2 += warp.size()) {
        float expv = expf(preatt_bth[t2] - maxval);
        expsum += expv;
    }

    expsum = cg::reduce(warp, expsum, cg::plus<float>{});

    float expsum_inv = expsum == 0.0f ? 0.0f : 1.0f / expsum;

    // pass 3: normalize to get the softmax is combined with the next loop to reduce memory round-trips
    for (int t2 = warp.thread_rank(); t2 <= t; t2 += warp.size()) {
        att_bth[t2] = expf(preatt_bth[t2] - maxval) * expsum_inv;
    }

    // pass 4: accumulate weighted values into the output of attention
    float* out_bth = out + b * T * C + t * C + h * hs;
    for (int i = warp.thread_rank(); i < hs; i += warp.size()) {
        float o = 0.f;
        for (int t2 = 0; t2 <= t; t2++) {
            const float* value_t2 = inp + b * T * C3 + t2 * C3 + h * hs + C * 2; // +C*2 because it's value
            float att_btht2 = att_bth[t2];
            o += att_btht2 * value_t2[i];
        }
        out_bth[i] = o;
    }
}

// ----------------------------------------------------------------------------
// kernel launcher

void attention_forward1(float* out, float* preatt, float* att,
                       const float* inp,
                       int B, int T, int C, int NH,
                       const int block_size) {
    // attention calculation
    int total_threads = B * NH * T * T;
    int num_blocks = ceil_div(total_threads, block_size);
    attention_query_key_kernel1<<<num_blocks, block_size>>>(preatt, inp, B, T, C, NH);
    // softmax and value accumulation
    total_threads = B * T * NH;
    num_blocks = ceil_div(total_threads, block_size);
    attention_softmax_kernel1<<<num_blocks, block_size>>>(att, preatt, B, T, NH);
    attention_value_kernel1<<<num_blocks, block_size>>>(out, att, inp, B, T, C, NH);
}


void attention_forward2(float* out,
                       const float* inp,
                       int B, int T, int C, int NH,
                       const int block_size) {
    // TODO there should be no mallocs inside any of these functions!
    // not fixing this because we don't intend to use attention_forward2,
    // it seems to be way too slow as is

    // these are hardcoded to 32 for now
    const int Bc = 32;
    const int Br = 32;
    // renaming these to be consistent with the kernel
    // const int B = B;
    const int nh = NH;
    const int N = T;
    const int d = C / NH;
    // more
    const int Tc = ceil((float) N / Bc);
    const int Tr = ceil((float) N / Br);
    const float softmax_scale = 1.0 / sqrt(d);
    // create some temporary memory
    float* l;
    float* m;
    cudaCheck(hipMalloc(&l, B * nh * N * sizeof(float)));
    cudaCheck(hipMalloc(&m, B * nh * N * sizeof(float)));
    cudaCheck(hipMemset(l, 0, B * nh * N * sizeof(float)));
    cudaCheck(hipMemset(m, -10000.0f, B * nh * N * sizeof(float)));

    // calculate SRAM size needed per block, ensure we have enough shared memory
    int col_tile_size = Bc * d;  // size of Kj, Vj
    int row_tile_size = Br * d;  // size of Qi
    const int sram_size =
        (2 * col_tile_size * sizeof(float))  // SRAM size for Kj, Vj
        + (row_tile_size * sizeof(float))  // SRAM size for Qi
        + (Bc * Br * sizeof(float));  // SRAM size for S
    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    if (sram_size > max_sram_size) {
        printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);
        printf("SRAM size exceeds maximum shared memory per block\n");
        printf("Try decreasing col_tile_size or row_tile_size further\n");
        exit(1);
    }

    // grid and block dims
    dim3 grid_dim(B, nh);  // batch_size x num_heads
    dim3 block_dim(Br);  // Br threads per block

    // okay so now, this kernel wants Q,K,V to all be of shape (B, nh, N, d)
    // but instead, we have a single tensor QKV (inp) of shape (B, N, 3, nh, d)
    // so we have to permute the tensor using a kernel with block_size
    float *q, *k, *v;
    cudaCheck(hipMalloc(&q, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&k, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&v, B * T * C * sizeof(float)));
    int total_threads = B * N * nh * d;
    int num_blocks = ceil_div(total_threads, block_size);
    permute_kernel<<<num_blocks, block_size>>>(q, k, v, inp, B, N, nh, d);

    // now actually call the flash attention kernel
    attention_forward_kernel2<<<grid_dim, block_dim, sram_size>>>(
        q, k, v,
        N, d, Tc, Tr, Bc, Br, softmax_scale,
        l, m, out
    );

    // out has shape (B, nh, N, d) but we need to unpermute it to (B, N, nh, d)
    unpermute_kernel<<<num_blocks, block_size>>>(out, q, B, N, nh, d);
    cudaCheck(hipMemcpy(out, q, B * T * C * sizeof(float), hipMemcpyDeviceToDevice));

    // free memory
    cudaCheck(hipFree(l));
    cudaCheck(hipFree(m));
    cudaCheck(hipFree(q));
    cudaCheck(hipFree(k));
    cudaCheck(hipFree(v));
}

void attention_forward3(float* out, float* vaccum, float* qkvr, float* preatt, float* att,
                       const float* inp,
                       int B, int T, int C, int NH,
                       const int block_size) {
    // inp is (B, T, 3C) QKV
    // preatt, att are (B, NH, T, T)
    // output is (B, T, C)
    int HS = C / NH; // head size

    // permute and separate inp from (B, T, 3, NH, HS) to 3X (B, NH, T, HS)
    float *q, *k, *v;
    q = qkvr + 0 * B * T * C;
    k = qkvr + 1 * B * T * C;
    v = qkvr + 2 * B * T * C;
    int total_threads = B * NH * T * HS;
    int num_blocks = ceil_div(total_threads, block_size);
    permute_kernel<<<num_blocks, block_size>>>(q, k, v, inp, B, T, NH, HS);

    // batched matrix multiply with cuBLAS
    const float alpha = 1.0f;
    const float beta = 0.0f;
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle,
                            HIPBLAS_OP_T, HIPBLAS_OP_N,
                            T, T, HS,
                            &alpha,
                            k, HS, T * HS,
                            q, HS, T * HS,
                            &beta,
                            preatt, T, T * T,
                            B * NH));

    // multiply all elements of preatt elementwise by scale
    float scale = 1.0f / sqrtf(HS);
    total_threads = B * NH * T * T;
    num_blocks = ceil_div(total_threads, block_size);
    scale_kernel<<<num_blocks, block_size>>>(preatt, scale, B, NH, T);

    // softmax. preatt is (B, NH, T, T) but we view it as (B * NH * T, T) and use the softmax kernel
    int softmax_block_size = 256;
    int grid_size = B * NH * T;
    size_t shared_mem_size = 2 * softmax_block_size / 32 * sizeof(float);
    softmax_forward_kernel4<<<grid_size, softmax_block_size, shared_mem_size>>>(att, preatt, B * NH * T, T);

    // new approach: first cuBLAS another batched matmul
    // y = att @ v # (B, nh, T, T) @ (B, nh, T, hs) -> (B, nh, T, hs)
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle,
                            HIPBLAS_OP_N, HIPBLAS_OP_N,
                            HS, T, T,
                            &alpha,
                            v, HS, T * HS,
                            att, T, T * T,
                            &beta,
                            vaccum, HS, T * HS,
                            B * NH));

    // now unpermute
    // y = y.transpose(1, 2).contiguous().view(B, T, C) # re-assemble all head outputs side by side
    num_blocks = ceil_div(B * T * C, block_size);
    unpermute_kernel<<<num_blocks, block_size>>>(vaccum, out, B, T, NH, HS);
}

void attention_forward4(float* out, float* vaccum, float* qkvr, float* preatt, float* att,
                        const float* inp,
                        int B, int T, int C, int NH,
                        const int block_size) {
    // inp is (B, T, 3C) QKV
    // preatt, att are (B, NH, T, T)
    // output is (B, T, C)
    int HS = C / NH; // head size

    // permute and separate inp from (B, T, 3, NH, HS) to 3X (B, NH, T, HS)
    float *q, *k, *v;
    q = qkvr + 0 * B * T * C;
    k = qkvr + 1 * B * T * C;
    v = qkvr + 2 * B * T * C;
    int total_threads = B * NH * T * HS;
    int num_blocks = ceil_div(total_threads, block_size);
    permute_kernel<<<num_blocks, block_size>>>(q, k, v, inp, B, T, NH, HS);

    // batched matrix multiply with cuBLAS
    const float alpha = 1.0f;
    const float beta = 0.0f;
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle,
                                     HIPBLAS_OP_T, HIPBLAS_OP_N,
                                     T, T, HS,
                                     &alpha,
                                     k, HS, T * HS,
                                     q, HS, T * HS,
                                     &beta,
                                     preatt, T, T * T,
                                     B * NH));

    // multiply all elements of preatt elementwise by scale
    float scale = 1.0 / sqrtf(HS);
    int softmax_block_size = 256;
    int grid_size = ceil_div(B * NH * T * 32, softmax_block_size);
    softmax_forward_kernel5<<<grid_size, softmax_block_size>>>(att, scale, preatt, B * NH, T);

    // new approach: first cuBLAS another batched matmul
    // y = att @ v # (B, nh, T, T) @ (B, nh, T, hs) -> (B, nh, T, hs)
    cublasCheck(hipblasSgemmStridedBatched(cublas_handle,
                                     HIPBLAS_OP_N, HIPBLAS_OP_N,
                                     HS, T, T,
                                     &alpha,
                                     v, HS, T * HS,
                                     att, T, T * T,
                                     &beta,
                                     vaccum, HS, T * HS,
                                     B * NH));

    // now unpermute
    // y = y.transpose(1, 2).contiguous().view(B, T, C) # re-assemble all head outputs side by side
    num_blocks = ceil_div(B * T * C, block_size);
    unpermute_kernel<<<num_blocks, block_size>>>(vaccum, out, B, T, NH, HS);
}

void attention_forward5(float* out, float* preatt, float* att,
                        const float* inp,
                        int B, int T, int C, int NH,
                        const int block_size) {
    // attention calculation
    int x_blocks = ceil_div(T, block_size / 32);
    attention_forward_fused1<<<dim3(x_blocks, NH, B), block_size>>>(out, preatt, att, inp, B, T, C, NH);
}

__global__ void flashattention(float *out, float *K, float *Q, float* V, float scaling, int T_r, int T_c, int seq_len)
{   // used by attention_forward6
    // define constants, could be adjusted for different hardware specs
    const int d = 64;
    const int B_c = 32;
    const int B_r = 32;
    const int BK = B_c;
    const int CACHE_Q = 0; // if 1 then cache Q in SMEM otherwise reload it over the tiles

    const int batch_offset = d * seq_len * blockIdx.x;
    const int TN = 4;
    const int TM = 4;
    const int num_tiles = d/32; // or d/BK, number of tiles that the attention computation is split into
    /*
    NOTE: all are fully loaded into shared memory SMEM, I think we should adjust this as second step to only loading it in tiles of B_r x 32 
    and iterating the mults over the 32 sized tiles this way we can have a larger d, while keeping occupancy high
    */
    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;

    // statically define in SMEM and still address it with indices
    //__shared__ float Q_i[B_r][d]; // uncomment only if you want to cache over full d (if CACHE_Q = 1)
    __shared__ float Q_i[B_r][BK]; // if you want to save SMEM loads and keep the full Q loaded then change this to [B_r][d]
    
    __shared__ float K_j[B_c][BK+1]; // reduce SMEM bank conflicts by adding 1 column as K will be loaded transposed!
    __shared__ float V_j[B_c][BK];
    
    // attention result
    __shared__ float S_i[B_r][B_c+1]; // reduce SMEM bank conflicts by adding 1 column (in the naive softmax part)
    
    const uint totalResultsBlocktile = B_r * B_c; // number of results to calculate per block
    const uint numThreadsBlocktile = totalResultsBlocktile / (TM * TN); // number of threads needed
    const int threadId_flat = threadIdx.y * blockDim.x + threadIdx.x; // flattened thread id  (used for coalesced loading of tiles)

    // each thread process one block at position:
    const int threadCol = threadId_flat % (B_c / TN);
    const int threadRow = threadId_flat / (B_c / TN);
        
    float l_i[TM]= {0.0};; // storing the intermediate sum of exponentials per row
    float m_i[TM]; // storing the intermediate max value of the rows
    float last_m[TM]; // storing the last max value of the rows
    float O_i[num_tiles * TN * TM] = {0.0}; // storing the intermediate results of the Outputs (each thread stores a chunk TM x TN per tile)
    
    // reset to min
    for (int ii = 0; ii < TM; ii++) {
        m_i[ii] = -INFINITY;
    }

    //WARNING: due to coalsecing I should probably add a second set of variables for using BK+1
    const uint strideK = numThreadsBlocktile / BK; // 64 / 64 = 1
    const uint innerRowK = threadId_flat / BK; // 0-63 / 64, 0000000000000...0
    const uint innerColK = threadId_flat % BK; // 0-63 % 64, 0123456789101112...63

    int id;
    // load Q_i, UNCOMMENT only if your Q is caching over full d
    const uint innerRowQ = threadId_flat / d; // 0-63 / 64, 0000000000000...0
    const uint innerColQ = threadId_flat % d; // 0-63 % 64, 0123456789012...63
    const uint nr_loads = B_r * d / numThreadsBlocktile;

    for (int t=0; t<nr_loads; t++){
      // need to load block of size B_r x d (64 x 64) with numThreadsBlocktile threads
      // if (blockIdx.y * B_r + innerRowQ) * d + innerColQ + t * numThreadsBlocktile / d
      id = (blockIdx.y * B_r + innerRowQ) * d + innerColQ + t * numThreadsBlocktile;
      // 4 x 4 then this is 5 thus 5/
      if (id < d*seq_len){
        Q_i[innerRowQ][innerColQ + t * numThreadsBlocktile] = Q[batch_offset + id];
      }
      else {
        Q_i[innerRowQ][innerColQ + t * numThreadsBlocktile] = 0.0;
      }
    }

    __syncthreads();

    // scratchpad register for register-tiling (coarsening of the matrix mults)
    float regM[TM] = {0.0};
    float regN[TN] = {0.0};

    for (int j = 0; j < T_c && j <= blockIdx.y ; j++) { // iterate of ver the chunks of K and V
        float threadResults[TM * TN] = {0.0}; // storing the intermediate outputs
        
        for (int t=0; t<num_tiles; t++){
            // load K_j and V_j, thread idx, idy loads idy,idx
            // we load a tile
            for (int i=0; i<B_r; i+=strideK){
                // load Q, K and V in tiles (for now we are loading the full V)
                if (not CACHE_Q){Q_i[innerRowK+i][innerColK] = Q[batch_offset + (innerRowK + blockIdx.y * B_r) * d  + i * d + innerColK + t * B_c];
                } // if you cache Q over whole d then remove this line
                id = (innerRowK + j * B_c) * d + i * d + innerColK + t * B_c;
                if (id < d*seq_len){
                    K_j[innerRowK+i][innerColK] = K[batch_offset + id];
                    //V_j[innerRowK+i][innerColK+t*B_c] = V[batch_offset + id];
                } else {
                    K_j[innerRowK+i][innerColK] = 0.0;
                    //V_j[innerRowK+i][innerColK+t*B_c] = 0.0;
                }
        
            }
            __syncthreads();
        
            for (int dd=0; dd<BK; dd++){ // load elements of Q_i and K_j^T into registers
                for (uint i = 0; i < TM; ++i) {
                    if (CACHE_Q){
                        regM[i] = Q_i[(threadRow * TM + i)][dd+t*BK]; // uncomment if you cache Q over full d
                    } else {
                        regM[i] = Q_i[(threadRow * TM + i)][dd];
                    }
                }
                for (uint i = 0; i < TN; ++i) {
                    regN[i] = K_j[threadCol * TN + i][dd];
                }
                for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
                    for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
                        threadResults[resIdxM * TN + resIdxN] += regM[resIdxM] * regN[resIdxN];
                    }
                }
            }
            __syncthreads();
        }
        

        // store the results in S_i, account for causal masking
        for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
            for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
                if (j*B_c + threadCol * TN + resIdxN <= blockIdx.y * B_r + threadRow * TM + resIdxM){
                    S_i[(threadRow * TM + resIdxM)][threadCol * TN + resIdxN] = threadResults[resIdxM * TN + resIdxN] *scaling;
                } else {
                    S_i[(threadRow * TM + resIdxM)][threadCol * TN + resIdxN] = -INFINITY;
                }      
            }
        }
        __syncthreads();

        for (int i=0;i<TM;++i){
            last_m[i] = m_i[i];
            float m = m_i[i];
            for (int jj = 0; jj < B_c; jj += 1) {
                if (m < S_i[threadRow*TM+i][jj]) {
                    m = S_i[threadRow*TM+i][jj];
                }
            }
            m_i[i] = m;
        }

        // 2) renormalize current O
        if (j > 0) {
            for (int t = 0; t < num_tiles; t++){
                for (int i=0;i<TM;++i){
                    for (int jj=0;jj<TN;++jj){
                        O_i[t*TN*TM + i*TN + jj] *= exp(last_m[i] - m_i[i]);
                    }
                }
            }
        }

        // 3) renormalize the sum l_i
        for (int i=0;i<TM;++i){
            l_i[i] *= exp(last_m[i] - m_i[i]);
        }

        // // 4) compute \exp(Q_iK^T_{j+1} - m^{j+1}) = \exp(S_i-m^{j+1}) // TODO: TO OPTIMIZE
        // for (int dd = 0; dd < B_c; dd++) {
        //     for (int ii = 0; ii < TN; ii++){ 
        //         // calculate new sum and load exp(Attention) weights
        //         //check whether thus is in range or not (if not we set it to 0)
        //         //if (idrow+ii < seq_len && idcol+dd < seq_len){
        //         regM[ii] = exp(S_i[threadRow*TM+ii][dd] - m_i[ii]);
        //         l_i[ii] += regM[ii];
        //     }
        //     for (int t = 0; t < num_tiles; t++){
        //         for (int ii=0;ii<TN;ii++){
        //             for (int jj=0;jj<TM;jj++){ // calculate output elements
        //                 regN[jj] = V_j[dd][t * B_c + threadCol * TN + jj];
        //                 O_i[t*TN*TM + ii*TM + jj] += regM[ii] * regN[jj];
        //             }
        //         }
        //     }
        // __syncthreads();
        // }


        for (int t = 0; t < num_tiles; t++){
            // load V
            __syncthreads();
            for (int i=0; i<B_r; i+=strideK){
                id = (innerRowK + j * B_c) * d + i * d + innerColK + t * B_c;
                if (id < d*seq_len){
                    V_j[innerRowK+i][innerColK] = V[batch_offset + id];
                } else {
                    V_j[innerRowK+i][innerColK] = 0.0;
                }
            }
            __syncthreads();

            for (int dd = 0; dd < B_c; dd++) {
                for (int ii = 0; ii < TN; ii++){
                    regM[ii] = exp(S_i[threadRow*TM+ii][dd] - m_i[ii]);
                    if (t==0){
                        l_i[ii] += regM[ii];
                    }
                    regN[ii] = V_j[dd][threadCol * TN + ii];
                }
                for (int ii=0;ii<TN;ii++){
                    for (int jj=0;jj<TM;jj++){ // calculate output elements
                        regN[jj] = V_j[dd][threadCol * TN + jj];
                        O_i[t*TN*TM + ii*TM + jj] += regM[ii] * regN[jj];
                    }
                }
            }
            __syncthreads();
        }
    }

    // normalize by the output sum and write to out matrix
    for (int t = 0; t < num_tiles; t++){
        for (int ii=0;ii<TM;ii++){
            for (int jj=0;jj<TN;jj++){
                if(blockIdx.y*B_r+threadRow*TM+ii < seq_len){
                    out[batch_offset + (blockIdx.y * B_r + threadRow*TM + ii) * d + t * B_c + threadCol*TN + jj] = O_i[t*TN*TM+ii*TM+jj] / l_i[ii];
                }
            }
        } 
    }
}

void attention_forward6(float* out,
                       const float* inp,
                       int B, int T, int C, int NH,
                       const int block_size) {
    // these are hardcoded to 32 for now
    const int B_r = 32;
    const int B_c = 32;
    // renaming these to be consistent with the kernel
    // const int B = B;
    const int nh = NH;
    const int N = T;
    const int d = C / NH;
    // more
    
    int TM = 4;
    int TN = 4;

    const float softmax_scale = 1.0 / sqrt(d);

    // calculate SRAM size needed per block, ensure we have enough shared memory
    int col_tile_size = B_r * d;  // size of Kj, Vj
    int row_tile_size = B_c * d;  // size of Qi
    const int sram_size =
        (col_tile_size * sizeof(float))  // SRAM size for Vj
        + (row_tile_size * sizeof(float))  // SRAM size for Qi
        + (B_c * (B_c+1) * sizeof(float)) // SRAM size for S
        + (B_c * (B_c+1) * sizeof(float)); // SRAM size for Kj, 

    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    if (sram_size > max_sram_size) {
        printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);
        printf("SRAM size exceeds maximum shared memory per block\n");
        printf("Try decreasing col_tile_size or row_tile_size further\n");
        exit(1);
    }

    // okay so now, this kernel wants Q,K,V to all be of shape (B, nh, N, d)
    // but instead, we have a single tensor QKV (inp) of shape (B, N, 3, nh, d)
    // so we have to permute the tensor using a kernel with block_size
    float *q, *k, *v;
    cudaCheck(hipMalloc(&q, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&k, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&v, B * T * C * sizeof(float)));

    dim3 blockDim(B_r/TN, B_c/TM);
    dim3 gridDim(B*nh, (N+B_r-1)/B_r);

    int total_threads = B * N * nh * d;
    int num_blocks = ceil_div(total_threads, block_size);
    
    permute_kernel<<<num_blocks, block_size>>>(q, k, v, inp, B, N, nh, d);

    // now actually call the flash attention kernel
    hipDeviceSynchronize();
    double start, end;
    start = getTimeStamp();
    flashattention<<<gridDim, blockDim>>>(out, k, q, v, softmax_scale, (N+B_r-1)/B_r, (N+B_c-1)/B_c, N);
    hipDeviceSynchronize();
    end = getTimeStamp();
    printf("Time taken for attention kernel: %f\n", end-start);

    // out has shape (B, nh, N, d) but we need to unpermute it to (B, N, nh, d)
    num_blocks = ceil_div(B * T * C, block_size);

    unpermute_kernel<<<num_blocks, block_size>>>(out, q, B, N, nh, d);
    hipDeviceSynchronize();
    cudaCheck(hipMemcpy(out, q, B * T * C * sizeof(float), hipMemcpyDeviceToDevice));
    hipDeviceSynchronize();
    // free memory
    cudaCheck(hipFree(q));
    cudaCheck(hipFree(k));
    cudaCheck(hipFree(v));
}


// kernel version dispatch
void attention_forward(int kernel_num,
                       float* out, float* vaccum, float* qkvr, float* preatt, float* att,
                       const float* inp,
                       int B, int T, int C, int NH,
                       const int block_size) {
    switch (kernel_num) {
        case 1:
            attention_forward1(out, preatt, att, inp, B, T, C, NH, block_size);
            break;
        case 2:
            attention_forward2(out, inp, B, T, C, NH, block_size);
            break;
        case 3:
            attention_forward3(out, vaccum, qkvr, preatt, att, inp, B, T, C, NH, block_size);
            break;
        case 4:
            attention_forward4(out, vaccum, qkvr, preatt, att, inp, B, T, C, NH, block_size);
            break;
        case 5:
            attention_forward5(out, preatt, att, inp, B, T, C, NH, block_size);
            break;
        case 6:
            attention_forward6(out, inp, B, T, C, NH, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}
// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    srand(0);

    int B = 6;
    int T = 4096;
    int C = 768;
    int NH = 12;

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));
    hipblasCreate(&cublas_handle);

    // create host memory of random numbers
    float* out = (float*)malloc(B * T * C * sizeof(float));
    float* preatt = (float*)malloc(B * NH * T * T * sizeof(float));
    float* att = (float*)malloc(B * NH * T * T * sizeof(float));
    float* inp = make_random_float(B * T * 3 * C);

    // move to GPU
    float* d_out;
    float* d_vaccum;
    float* d_qkvr;
    float* d_preatt;
    float* d_att;
    float* d_inp;
    cudaCheck(hipMalloc(&d_out, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_vaccum, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_qkvr, B * T * 3 * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_preatt, B * NH * T * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_att, B * NH * T * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * 3 * C * sizeof(float)));
    cudaCheck(hipMemcpy(d_inp, inp, B * T * 3 * C * sizeof(float), hipMemcpyHostToDevice));

    // read kernel_num from command line
    int kernel_num = 1;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);
    int block_sizes[] = {32, 64, 128, 256, 512};

    // first check the correctness of the kernel
    attention_forward_cpu(out, preatt, att, inp, B, T, C, NH);
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        printf("Checking block size %d.\n", block_size);
        attention_forward(kernel_num, d_out, d_vaccum, d_qkvr, d_preatt, d_att, d_inp, B, T, C, NH, block_size);
        // all kernels should produce the correct output out
        validate_result(d_out, out, "out", B * T * C, 1e-4f);
        // but as for preatt and att, things get a bit more complicated:
        if (kernel_num != 2 && kernel_num != 6) {
            // kernel 2 (knowingly) fails att/preatt because it uses a different algorithm
            // that estimates the softmax online and never materializes preatt/att
            validate_result(d_att, att, "att", B * NH * T * T, 1e-4f);
        }
        if (kernel_num != 2 && kernel_num != 4 && kernel_num != 5 && kernel_num != 6) {
            // kernel 4 (knowingly) fails preatt because it fuses the scale normalization
            // into the softmax, so preatt is off by 1.0f / sqrt(HS)
            // but att and out (checked below) should match.
            validate_result(d_preatt, preatt, "preatt", B * NH * T * T, 1e-4f);
        }
    }
    printf("All results match. Starting benchmarks.\n\n");

    // benchmark speed of the kernel
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        int repeat_times = 100;

        float elapsed_time = benchmark_kernel(repeat_times, attention_forward,
                                              kernel_num, d_out, d_vaccum, d_qkvr, d_preatt, d_att, d_inp,
                                              B, T, C, NH, block_size);

        printf("block_size %4d | time %f ms\n", block_size, elapsed_time);
    }

    // free memory
    free(out);
    free(preatt);
    free(att);
    free(inp);
    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_vaccum));
    cudaCheck(hipFree(d_qkvr));
    cudaCheck(hipFree(d_preatt));
    cudaCheck(hipFree(d_att));
    cudaCheck(hipFree(d_inp));
    hipblasDestroy(cublas_handle);

    return 0;
}